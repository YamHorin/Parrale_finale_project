#include <stdio.h>
#include <string.h>
#include <ctype.h>
#include <hip/hip_runtime.h> // Include CUDA runtime header

#define BLOCK_DIM 1024 // number of threads in a block
#define MAX_STRING_SIZE 3000
#define MATRIX_SIZE 26

__device__ char gpu_toupper(char c)
{
    if (c >= 'a' && c <= 'A')
        return c - ('a' - 'A');
    return c;
}

__global__ void Mutanat_SquenceKernel(char *str, int k, int size_str)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= k && tid <= size_str)
    {
        if (gpu_toupper(str[tid]) >= 'Z')
        {
            str[tid] = 'A';
        }
        if (tid == size_str)
        {
            str[tid] = '\0';
        }
        else
        {
            str[tid] = gpu_toupper(str[tid] + 1);
        }
    }
}

int Mutanat_Squence_cuda(char *str, int k, int size_str)
{
    char *d_str;
    int strSize = size_str + 1; // Include space for '\0'

    // Allocate memory on the GPU
    hipError_t cudaStatus = hipMalloc((void **)&d_str, strSize * sizeof(char));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        return -1;
    }

    // Copy the input string to the GPU
    cudaStatus = hipMemcpy(d_str, str, strSize * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_str);
        return -1;
    }

    // Calculate the number of threads per block and the number of blocks
    int threadsPerBlock = 256;
    int numBlocks = (strSize + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    Mutanat_SquenceKernel<<<numBlocks, threadsPerBlock>>>(d_str, k, size_str);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_str);
        return -1;
    }

    // Copy the result back from the GPU to the CPU
    cudaStatus = hipMemcpy(str, d_str, strSize * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy (device to host) failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_str);
        return -1;
    }

    // Free the GPU memory
    hipFree(d_str);

    return 0; // Return 0 on success
}
