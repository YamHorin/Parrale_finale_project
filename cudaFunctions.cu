#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstring>
#include <ctype.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "struct.h"


__device__ int device_strlen(const char *str)
{
    int length = 0;
    while (str[length] != '\0')
    {
        length++;
    }
    return length;
}

__device__ void device_strncpy(char *dest, const char *src, int n)
{
    for (int i = 0; i < n; i++)
    {
        dest[i] = src[i];
    }
    dest[n] = '\0';
}

__device__ char gpu_toupper(char c)
{
    if (c >= 'a' && c <= 'A')
        return c - ('a' - 'A');
    return c;
}
__global__ void caculate_result_without_matrix(char *str_to_check, char *first_str, int size_second_str, int *result, int off_set, int k)
{
    __shared__ int r;
    int value;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid == 0)
        r = 0;
    __syncthreads();
    if (tid < size_second_str)
    {
        int i = tid;
        if (tid >= k)
        {

            int x = gpu_toupper(first_str[i + off_set]) - 'A';
            int y = (gpu_toupper(str_to_check[i]) + 1) - 'A';
            value = (x == y);
        }
        else
        {
            int x = gpu_toupper(first_str[i + off_set]) - 'A';
            int y = (gpu_toupper(str_to_check[i])) - 'A';
            value = (x == y);
        }
        atomicAdd(&r, value);
    }
    __syncthreads();
    if (tid == 0)
        *result = r;
}
__global__ void caculate_result(char *str_to_check, char *first_str, int size_second_str, int *result, int off_set, int *matrix, int k)
{
    __shared__ int r;
    int value;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid == 0)
        r = 0;
    __syncthreads();
    if (tid < size_second_str)
    {
        int i = tid;
        if (tid >= k)
        {

            int x = gpu_toupper(first_str[i + off_set]) - 'A';
            int y = (gpu_toupper(str_to_check[i]) + 1) - 'A';
            value = matrix[x * MATRIX_SIZE + y];
        }
        else
        {
            int x = gpu_toupper(first_str[i + off_set]) - 'A';
            int y = (gpu_toupper(str_to_check[i])) - 'A';
            value = matrix[x * MATRIX_SIZE + y];
        }
        atomicAdd(&r, value);
    }
    __syncthreads();
    if (tid == 0)
        *result = r;
}

int caculate_cuda(const char *str_to_check, const char *first_str, int matrix[MATRIX_SIZE][MATRIX_SIZE] , int my_rank)
{

    // Calculate the lengths of the strings
    int size_str_to_check = strlen(str_to_check);
    int size_first_str = strlen(first_str);

    // Check if the lengths exceed the maximum allowed size
    if (size_str_to_check >= MAX_STRING_SIZE || size_first_str >= MAX_STRING_SIZE)
    {
        printf("Error: String length exceeds MAX_STRING_SIZE.\n");
        return -1; // or any appropriate error code
    }

    // Allocate memory on the GPU
    struct score_alignment localMax;
    char *d_str_to_check, *d_first_str;
    int *d_matrix, *dev_result;

    hipError_t err1 = hipMalloc((void **)&d_str_to_check, MAX_STRING_SIZE);

    hipError_t err2 = hipMalloc((void **)&d_first_str, MAX_STRING_SIZE);

    hipError_t err3 = hipMalloc((void **)&d_matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(int));

    hipError_t err4 = hipMalloc((void **)&dev_result, sizeof(int));

    if (err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess)
    {
        fprintf(stderr, "CUDA  malloc error\n");
        exit(1);
    }
    // Copy data from host to device
    err1 = hipMemcpy(d_str_to_check, str_to_check, size_str_to_check + 1, hipMemcpyHostToDevice);

    err2 = hipMemcpy(d_first_str, first_str, size_first_str + 1, hipMemcpyHostToDevice);

    err3 = hipMemcpy(d_matrix, matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(int), hipMemcpyHostToDevice);
    if (err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess)
    {
        fprintf(stderr, "CUDA  memcpy 1-3 error\n");
        exit(1);
    }

    int threadsPerBlock = 256;
    int result = 0;
    int blocksPerGrid = (size_str_to_check > MAX_STRING_SIZE) ? size_str_to_check / threadsPerBlock : 1;
    int max_score = 0;
    int sqn_taries = (size_str_to_check < size_first_str) ? (size_first_str - size_str_to_check) : (size_str_to_check - size_first_str);

    for (int off_set = 0; off_set <= sqn_taries; off_set++)
    {
        for (int k = 0; k < size_str_to_check; k++)
        {
            caculate_result<<<blocksPerGrid, threadsPerBlock>>>(
                d_str_to_check, d_first_str, size_str_to_check,
                dev_result, off_set, d_matrix, k);
            err4 = hipMemcpy(&result, dev_result, sizeof(int), hipMemcpyDeviceToHost);
            if (err4 != hipSuccess)
            {
                fprintf(stderr, "CUDA  memcpy 4 error\n");
                exit(1);
            }
            if (result >= max_score)
            {
                max_score = result;
                localMax.score = result;
                localMax.K = k;
                localMax.off_set = off_set;
            }
        }
    }

    hipFree(d_str_to_check);
    hipFree(d_first_str);
    hipFree(d_matrix);
    hipFree(dev_result);
    // Print the result
    printf("We found that the max score alignment %d is from K - %d and off set - %d\n", localMax.score, localMax.K, localMax.off_set);

    return 0;
}

int caculate_cuda_without_matrix(const char *str_to_check, const char *first_str , int my_rank)
{
    // Calculate the lengths of the strings
    int size_str_to_check = strlen(str_to_check);
    int size_first_str = strlen(first_str);

    // Check if the lengths exceed the maximum allowed size
    if (size_str_to_check >= MAX_STRING_SIZE || size_first_str >= MAX_STRING_SIZE)
    {
        printf("Error: String length exceeds MAX_STRING_SIZE.\n");
        return -1; // or any appropriate error code
    }

    // Allocate memory on the GPU
    struct score_alignment localMax;
    char *d_str_to_check, *d_first_str;
    int *dev_result;
    hipError_t err1 = hipMalloc((void **)&d_str_to_check, MAX_STRING_SIZE);

    hipError_t err2 = hipMalloc((void **)&d_first_str, MAX_STRING_SIZE);

    hipError_t err3 = hipMalloc((void **)&dev_result, sizeof(int));

    if (err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess)
    {
        fprintf(stderr, "CUDA  malloc  error\n");
        exit(1);
    }
    // Copy data from host to device
    err1 = hipMemcpy(d_str_to_check, str_to_check, size_str_to_check + 1, hipMemcpyHostToDevice);

    err2 = hipMemcpy(d_first_str, first_str, size_first_str + 1, hipMemcpyHostToDevice);

    if (err1 != hipSuccess || err2 != hipSuccess)
    {
        fprintf(stderr, "CUDA   memcpy  error\n");
        exit(1);
    }
    int threadsPerBlock = 256;
    int result = 0;
    int blocksPerGrid = (size_str_to_check > MAX_STRING_SIZE) ? size_str_to_check / threadsPerBlock : 1;
    int max_score = 0;
    int sqn_taries = (size_str_to_check < size_first_str) ? (size_first_str - size_str_to_check) : (size_str_to_check - size_first_str);

    for (int off_set = 0; off_set <= sqn_taries; off_set++)
    {
        for (int k = 0; k < size_str_to_check; k++)
        {
            caculate_result_without_matrix<<<blocksPerGrid, threadsPerBlock>>>(
                d_str_to_check, d_first_str, size_str_to_check,
                dev_result, off_set, k);
            err3 = hipMemcpy(&result, dev_result, sizeof(int), hipMemcpyDeviceToHost);
            if (err3 != hipSuccess)
            {
                fprintf(stderr, "CUDA  memcpy 2 error\n");
                exit(1);
            }
            if (result >= max_score)
            {
                max_score = result;
                localMax.score = result;
                localMax.K = k;
                localMax.off_set = off_set;
            }
        }
    }

    hipFree(d_str_to_check);
    hipFree(d_first_str);
    hipFree(dev_result);
    // Print the result
    printf("We found that the max score alignment %d is from K - %d and off set - %d\n", localMax.score, localMax.K, localMax.off_set);

    return 0;
}
