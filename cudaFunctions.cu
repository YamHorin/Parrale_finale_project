#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <ctype.h>
#include <hip/hip_runtime.h> // Include CUDA runtime header

#define BLOCK_DIM 1024 // number of threads in a block
#define MAX_STRING_SIZE 3000
#define MATRIX_SIZE 26


__device__ char gpu_toupper(char c)
{
    if (c>='a' && c<='A')
        return c-('a'-'A');
    return c;
}
// __device__ int getScoreFromMatrix(char a, char b) {
//     int x = a - 'A';
//     int y = b - 'A';
//     return matrix_caculate[x * MATRIX_SIZE + y]; // Assuming matrix_caculate is a 1D array representation of a 2D matrix.
// }


// __device__ void scan_plus(int *array, int size)
// {
//    for (unsigned int stride=1; stride <= size/2; stride *= 2) {
//         int v;
//         if (threadIdx.x >= stride) {
//             v = array[threadIdx.x - stride];
//         }
//         __syncthreads(); /* wait until all threads finish reading 
// 		                    an element */

//         if (threadIdx.x >= stride)
//             array[threadIdx.x] += v;

//         __syncthreads(); /* wait until all threads finish updating an
// 		                    element */
//      }
     
// } // scan_plus




// __global__ void caculateWithMatrix(const char  *s1, const char *s2, int n2,  int *result , int off_set)
// {
//      __shared__ int r;
//      int value;
//      int tid = threadIdx.x;
//     if(tid == 0) r = 0;
//     __syncthreads();
//     if ((tid +off_set)< n2)
//     {
//         value = (getScoreFromMatrix (s1[tid +off_set] ,s2[tid])); 
//         atomicAdd(&r , value);

//     }
//      //scan_plus(flags, BLOCK_DIM);
//      //if (tid  == BLOCK_DIM-1; 
//      __syncthreads();
//      if (tid == 0)
//         *result = r;

// }
// __global__ void caculate(const char  *s1, const char *s2, int n2,  int *result , int off_set)
// {
//     // __shared__ int flags[BLOCK_DIM];
//      __shared__ int r;
//      int value;
//      int tid = threadIdx.x;
//     if(tid == 0) r = 0;
//     __syncthreads();
//     if ((tid +off_set) < n2)
//     {
//         value = (s1[tid +off_set] == s2[tid]); 
//         atomicAdd(&r , value);

//     }
        

//      //scan_plus(flags, BLOCK_DIM);
//      //if (tid  == BLOCK_DIM-1; 
//      __syncthreads();
//      if (tid == 0)
//         *result = r;

// }


// int computeOnGPU(const char *s2 , int off_set) {
//     char *dev_s2;
//     int *dev_result;
//      // null byte at the end is also counted
//     int n2 = strlen(s2);
//     // allocate the memory on the GPU
//     hipError_t err2 = hipMalloc((void**)&dev_s2, n2);
//     if (err2 != hipSuccess)
//     {
//         fprintf(stderr, "CUDA  1 error\n");
//         exit(1);
//     }
//     hipError_t err3 =  hipMalloc((void**)&dev_result, sizeof(int));
//     if (err3 != hipSuccess)
//     {
//         fprintf(stderr, "CUDA 2 error\n");
//         exit(1);
//     }
//     hipMemcpy(dev_s2, s2, n2, hipMemcpyHostToDevice);
//     int threadsPerBlock = BLOCK_DIM;
//     int numOfBlocks = 1;
//     //if strlen <1024
//     caculate<<<numOfBlocks, threadsPerBlock>>>(first_str,dev_s2, n2, dev_result ,off_set);
//     err2 = hipGetLastError();
//     if (err2 != hipSuccess)
//     {
//         fprintf(stderr , "kerner lanch error\n");
//         exit(1);
//     }
//     // copy the result back from the GPU to the CPU
//     int result;
//     err2 = hipMemcpy(&result, dev_result, sizeof(int), hipMemcpyDeviceToHost);		
//     if (err2 != hipSuccess)
//     {
//         fprintf(stderr , "kerner lanch error");
//         exit(1);
//     }
//     // free memory on the GPU side
//     hipFree(dev_s2);
//     hipFree(dev_result);
//     return result;
    
// }
// int computeOnGPUWithMatrix( const char *s2 ,const int matrix[MATRIX_SIZE][MATRIX_SIZE] , int off_set)
// {
//     hipMemcpyToSymbol(HIP_SYMBOL(matrix_caculate), matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
//     char *dev_s2;
//     int *dev_result;
//     int n2 = strlen(s2);

//     // allocate the memory on the GPU
//     hipError_t err2 = hipMalloc((void**)&dev_s2, n2);
//     hipError_t err3 =  hipMalloc((void**)&dev_result, sizeof(int));
//     if(err2 != hipSuccess || err3 != hipSuccess) {
//         fprintf(stderr, "CUDA error\n");
//         exit(1);
//     }
//     hipMemcpy(dev_s2, s2, n2, hipMemcpyHostToDevice);
    
//     int threadsPerBlock = BLOCK_DIM;
//     int numOfBlocks = n2/BLOCK_DIM;
 
//     caculateWithMatrix<<<numOfBlocks, threadsPerBlock>>>(first_str, dev_s2, n2, dev_result ,off_set);
 
//     // copy the result back from the GPU to the CPU
//     int result=0;
//     hipMemcpy(&result, dev_result, sizeof(int), hipMemcpyDeviceToHost);		
	    
//     // free memory on the GPU side
//     hipFree(dev_s2);
//     hipFree(dev_result);

//     return result;
// }



// char *offsetFirstStr(int offset , int lenght)
// {
//     char *result, *returnStr;
//     hipMalloc((void **)&result, lenght);
//     int threadsPerBlock = BLOCK_DIM;
//     int numOfBlocks = 1;
//     change_offset<<<numOfBlocks, threadsPerBlock>>>(result, offset);
//     returnStr = (char *)malloc(lenght * sizeof(char));
//     hipMemcpy(returnStr, result, lenght * sizeof(char), hipMemcpyDeviceToHost);
//     hipFree(result);
//     return returnStr;
// }


__global__ void Mutanat_SquenceKernel(char* str, int k, int size_str) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= k && tid <= size_str) {
        if (toupper(str[tid]) >= 'Z') {
            str[tid] = 'A';
        }
        if (tid == size_str) {
            str[tid] = '\0';
        }
        else {
            str[tid] = toupper(str[tid + 1]);
        }
    }
}

int Mutanat_Squence(char* str, int k, int size_str) {
    char* d_str;
    int strSize = size_str + 1; // Include space for '\0'

    // Allocate memory on the GPU
    hipError_t cudaStatus = hipMalloc((void**)&d_str, strSize * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
        return -1;
    }

    // Copy the input string to the GPU
    cudaStatus = hipMemcpy(d_str, str, strSize * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_str);
        return -1;
    }

    // Calculate the number of threads per block and the number of blocks
    int threadsPerBlock = 256;
    int numBlocks = (strSize + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    Mutanat_SquenceKernel<<<numBlocks, threadsPerBlock>>>(d_str, k, size_str);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_str);
        return -1;
    }

    // Copy the result back from the GPU to the CPU
    cudaStatus = hipMemcpy(str, d_str, strSize * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy (device to host) failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(d_str);
        return -1;
    }

    // Free the GPU memory
    hipFree(d_str);

    return 0; // Return 0 on success
}
