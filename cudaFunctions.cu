
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <ctype.h>

#define MATRIX_SIZE 26
#define BLOCK_DIM 1024 // number of threads in a block

__device__ int matrix_caculate[MATRIX_SIZE*MATRIX_SIZE];





__device__ int getScoreFromMatrix(char a, char b) {
    int x = a - 'A';
    int y = b - 'A';
    //printf("%d\n",matrix_caculate[x * MATRIX_SIZE + y]);
    return matrix_caculate[x * MATRIX_SIZE + y]; // Assuming matrix_caculate is a 1D array representation of a 2D matrix.
}
// __device__ char gpu_toupper(char c) {
//     if (c >= 'a' && c <= 'z') {
//         return c - ('a' - 'A');
//     }
//     return c;
// }

__device__ void scan_plus(int *array, int size)
{
   for (unsigned int stride=1; stride <= size/2; stride *= 2) {
        int v;
        if (threadIdx.x >= stride) {
            v = array[threadIdx.x - stride];
        }
        __syncthreads(); /* wait until all threads finish reading 
		                    an element */

        if (threadIdx.x >= stride)
            array[threadIdx.x] += v;

        __syncthreads(); /* wait until all threads finish updating an
		                    element */
     }
     
} // scan_plus



__global__ void caculateWithMatrix(const char  *s1, int n1, const char *s2, int n2,  int *result)
{
     __shared__ int r;
     int value;
     int tid = threadIdx.x;
    if(tid == 0) r = 0;
    __syncthreads();
    if (tid < n1 && tid <n2)
    {
        value = (getScoreFromMatrix (s1[tid] ,s2[tid])); 
        atomicAdd(&r , value);

    }
        

     //scan_plus(flags, BLOCK_DIM);
     //if (tid  == BLOCK_DIM-1; 
     __syncthreads();
     if (tid == 0)
        *result = r;

}
__global__ void caculate(const char  *s1, int n1, const char *s2, int n2,  int *result)
{
    // __shared__ int flags[BLOCK_DIM];
     __shared__ int r;
     int value;
     int tid = threadIdx.x;
    if(tid == 0) r = 0;
    __syncthreads();
    if (tid < n1 && tid <n2)
    {
        value = (s1[tid] == s2[tid]); 
        atomicAdd(&r , value);

    }
        

     //scan_plus(flags, BLOCK_DIM);
     //if (tid  == BLOCK_DIM-1; 
     __syncthreads();
     if (tid == 0)
        *result = r;

}

// returns 0 if successful, otherwise returns 1
int computeOnGPU(const char  *s1, const char *s2) {
    char *dev_s1, *dev_s2;
    int *dev_result;
    
    int n1 = strlen(s1); // null byte at the end is also counted
    int n2 = strlen(s2);
    // allocate the memory on the GPU

    hipError_t err1 = hipMalloc((void**)&dev_s1, n1);
    if (err1 != hipSuccess)
    {
        fprintf(stderr, "CUDA 1 error\n");
        exit(1);
    }
    hipError_t err2 = hipMalloc((void**)&dev_s2, n2);
    if (err2 != hipSuccess)
    {
        fprintf(stderr, "CUDA  2 error\n");
        exit(1);
    }
    hipError_t err3 =  hipMalloc((void**)&dev_result, sizeof(int));
    if (err3 != hipSuccess)
    {
        fprintf(stderr, "CUDA 3 error\n");
        exit(1);
    }
    hipMemcpy(dev_s1, s1, n1, hipMemcpyHostToDevice);
    hipMemcpy(dev_s2, s2, n2, hipMemcpyHostToDevice);
    int threadsPerBlock = BLOCK_DIM;
    int numOfBlocks = 1;
    //if strlen <1024
    caculate<<<numOfBlocks, threadsPerBlock>>>(dev_s1, n1, dev_s2, n2, dev_result);
    err1 = hipGetLastError();
    if (err1 != hipSuccess)
    {
        fprintf(stderr , "kerner lanch error\n");
        exit(1);
    }
    // copy the result back from the GPU to the CPU
    int result;
    err1 = hipMemcpy(&result, dev_result, sizeof(int), hipMemcpyDeviceToHost);		
    if (err1 != hipSuccess)
    {
        fprintf(stderr , "kerner lanch error");
        exit(1);
    }
    // free memory on the GPU side
    hipFree(dev_s1);
    hipFree(dev_s2);
    hipFree(dev_result);
    return result;
    
}
int computeOnGPUWithMatrix(const char  *s1, const char *s2 ,const int matrix[MATRIX_SIZE][MATRIX_SIZE])
{
    hipMemcpyToSymbol(HIP_SYMBOL(matrix_caculate), matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
    char *dev_s1, *dev_s2;
    int *dev_result;
    int n1 = strlen(s1)+1; // null byte at the end is also counted
    int n2 = strlen(s2)+1;

    // allocate the memory on the GPU
    hipError_t err1 = hipMalloc((void**)&dev_s1, n1);
    hipError_t err2 = hipMalloc((void**)&dev_s2, n2);
    hipError_t err3 =  hipMalloc((void**)&dev_result, sizeof(int));
    if(err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess) {
        fprintf(stderr, "CUDA error\n");
        exit(1);
    }
    hipMemcpy(dev_s1, s1, n1, hipMemcpyHostToDevice);
    hipMemcpy(dev_s2, s2, n2, hipMemcpyHostToDevice);
    
    int threadsPerBlock = BLOCK_DIM;
    int numOfBlocks = 1;
 
    caculateWithMatrix<<<numOfBlocks, threadsPerBlock>>>(dev_s1, n1, dev_s2, n2, dev_result);
 
    // copy the result back from the GPU to the CPU
    int result=0;
    hipMemcpy(&result, dev_result, sizeof(int), hipMemcpyDeviceToHost);		
	    
    // free memory on the GPU side
    hipFree(dev_s1);
    hipFree(dev_s2);
    hipFree(dev_result);

    return result;
}



