#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <ctype.h>
#include <hip/hip_runtime.h> // Include CUDA runtime header

#define BLOCK_DIM 1024 // number of threads in a block
#define MAX_STRING_SIZE 3000
#define MATRIX_SIZE 26

__device__ int matrix_caculate[MATRIX_SIZE * MATRIX_SIZE];
__device__ char Str_to_check[MAX_STRING_SIZE];
__device__ char first_str[MAX_STRING_SIZE];
__device__ int length_first_str;

__device__ char gpu_toupper(char c)
{
    if (c>='a' && c<='A')
        return c-('a'-'A');
    return c;
}
__device__ int getScoreFromMatrix(char a, char b) {
    int x = a - 'A';
    int y = b - 'A';
    return matrix_caculate[x * MATRIX_SIZE + y]; // Assuming matrix_caculate is a 1D array representation of a 2D matrix.
}


__device__ void scan_plus(int *array, int size)
{
   for (unsigned int stride=1; stride <= size/2; stride *= 2) {
        int v;
        if (threadIdx.x >= stride) {
            v = array[threadIdx.x - stride];
        }
        __syncthreads(); /* wait until all threads finish reading 
		                    an element */

        if (threadIdx.x >= stride)
            array[threadIdx.x] += v;

        __syncthreads(); /* wait until all threads finish updating an
		                    element */
     }
     
} // scan_plus




__global__ void caculateWithMatrix(const char  *s1, const char *s2, int n2,  int *result , int off_set)
{
     __shared__ int r;
     int value;
     int tid = threadIdx.x;
    if(tid == 0) r = 0;
    __syncthreads();
    if ((tid +off_set)< n2)
    {
        value = (getScoreFromMatrix (s1[tid +off_set] ,s2[tid])); 
        atomicAdd(&r , value);

    }
     //scan_plus(flags, BLOCK_DIM);
     //if (tid  == BLOCK_DIM-1; 
     __syncthreads();
     if (tid == 0)
        *result = r;

}
__global__ void caculate(const char  *s1, const char *s2, int n2,  int *result , int off_set)
{
    // __shared__ int flags[BLOCK_DIM];
     __shared__ int r;
     int value;
     int tid = threadIdx.x;
    if(tid == 0) r = 0;
    __syncthreads();
    if ((tid +off_set) < n2)
    {
        value = (s1[tid +off_set] == s2[tid]); 
        atomicAdd(&r , value);

    }
        

     //scan_plus(flags, BLOCK_DIM);
     //if (tid  == BLOCK_DIM-1; 
     __syncthreads();
     if (tid == 0)
        *result = r;

}


int computeOnGPU(const char *s2 , int off_set) {
    char *dev_s2;
    int *dev_result;
     // null byte at the end is also counted
    int n2 = strlen(s2);
    // allocate the memory on the GPU
    hipError_t err2 = hipMalloc((void**)&dev_s2, n2);
    if (err2 != hipSuccess)
    {
        fprintf(stderr, "CUDA  1 error\n");
        exit(1);
    }
    hipError_t err3 =  hipMalloc((void**)&dev_result, sizeof(int));
    if (err3 != hipSuccess)
    {
        fprintf(stderr, "CUDA 2 error\n");
        exit(1);
    }
    hipMemcpy(dev_s2, s2, n2, hipMemcpyHostToDevice);
    int threadsPerBlock = BLOCK_DIM;
    int numOfBlocks = 1;
    //if strlen <1024
    caculate<<<numOfBlocks, threadsPerBlock>>>(first_str,dev_s2, n2, dev_result ,off_set);
    err2 = hipGetLastError();
    if (err2 != hipSuccess)
    {
        fprintf(stderr , "kerner lanch error\n");
        exit(1);
    }
    // copy the result back from the GPU to the CPU
    int result;
    err2 = hipMemcpy(&result, dev_result, sizeof(int), hipMemcpyDeviceToHost);		
    if (err2 != hipSuccess)
    {
        fprintf(stderr , "kerner lanch error");
        exit(1);
    }
    // free memory on the GPU side
    hipFree(dev_s2);
    hipFree(dev_result);
    return result;
    
}
int computeOnGPUWithMatrix( const char *s2 ,const int matrix[MATRIX_SIZE][MATRIX_SIZE] , int off_set)
{
    hipMemcpyToSymbol(HIP_SYMBOL(matrix_caculate), matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
    char *dev_s2;
    int *dev_result;
    int n2 = strlen(s2);

    // allocate the memory on the GPU
    hipError_t err2 = hipMalloc((void**)&dev_s2, n2);
    hipError_t err3 =  hipMalloc((void**)&dev_result, sizeof(int));
    if(err2 != hipSuccess || err3 != hipSuccess) {
        fprintf(stderr, "CUDA error\n");
        exit(1);
    }
    hipMemcpy(dev_s2, s2, n2, hipMemcpyHostToDevice);
    
    int threadsPerBlock = BLOCK_DIM;
    int numOfBlocks = n2/BLOCK_DIM;
 
    caculateWithMatrix<<<numOfBlocks, threadsPerBlock>>>(first_str, dev_s2, n2, dev_result ,off_set);
 
    // copy the result back from the GPU to the CPU
    int result=0;
    hipMemcpy(&result, dev_result, sizeof(int), hipMemcpyDeviceToHost);		
	    
    // free memory on the GPU side
    hipFree(dev_s2);
    hipFree(dev_result);

    return result;
}
void getStrToCheck(char *s1, int n1)
{
    hipMemcpyToSymbol(HIP_SYMBOL(Str_to_check), s1, n1 * sizeof(char));
}

void getFirstStr(char *s1, int n1)
{
    hipMemcpyToSymbol(HIP_SYMBOL(first_str), s1, n1 * sizeof(char));
    hipMemcpyToSymbol(HIP_SYMBOL(length_first_str), &n1, sizeof(int));
}

__global__ void change_offset(char *str, int offset)
{
    int tid = threadIdx.x;
    if ((tid + offset) < length_first_str)
    {
        str[tid] = first_str[tid + offset];
    }
    if (tid == length_first_str)
    {
        str[tid] = '\0';
    }
}

char *offsetFirstStr(int offset , int lenght)
{
    char *result, *returnStr;
    hipMalloc((void **)&result, lenght);
    int threadsPerBlock = BLOCK_DIM;
    int numOfBlocks = 1;
    change_offset<<<numOfBlocks, threadsPerBlock>>>(result, offset);
    returnStr = (char *)malloc(lenght * sizeof(char));
    hipMemcpy(returnStr, result, lenght * sizeof(char), hipMemcpyDeviceToHost);
    hipFree(result);
    return returnStr;
}

__global__ void change_mutant_squence(char *str, int k , int size_str)
{
    int tid = threadIdx.x;
    if (tid<=size_str && tid >= k)
    {
        if (gpu_toupper(str[tid])>='Z')
            str[tid] = 'A';
        if (tid==size_str)
            str[tid] = '\0';
        else
            str[tid] = gpu_toupper(Str_to_check[tid]+1);
    }
}

char[MAX_STRING_SIZE] Mutanat_Squence_cuda(int k , int size_str)
{
    char *result, 
    char [MAX_STRING_SIZE] returnStr = NULL;
    hipMalloc((void **)&result, size_str);
    int threadsPerBlock = BLOCK_DIM;
    int numOfBlocks = 1;
    change_mutant_squence<<<numOfBlocks , threadsPerBlock>>>(result , k , size_str);
   
    hipMemcpy(returnStr, result, size_str * sizeof(char), hipMemcpyDeviceToHost);
    hipFree(result);
    return returnStr;

}