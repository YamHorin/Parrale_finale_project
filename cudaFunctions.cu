#include <stdio.h>
#include <string.h>
#include <ctype.h>
#include <hip/hip_runtime.h> // Include CUDA runtime header

#define MATRIX_SIZE 26
#define BLOCK_DIM 1024 // number of threads in a block

__device__ int matrix_caculate[MATRIX_SIZE * MATRIX_SIZE];
__device__ char first_str[BLOCK_DIM];
__device__ int length_first_str;


__device__ int getScoreFromMatrix(char a, char b) {
    int x = a - 'A';
    int y = b - 'A';
    return matrix_caculate[x * MATRIX_SIZE + y]; // Assuming matrix_caculate is a 1D array representation of a 2D matrix.
}


__device__ void scan_plus(int *array, int size)
{
   for (unsigned int stride=1; stride <= size/2; stride *= 2) {
        int v;
        if (threadIdx.x >= stride) {
            v = array[threadIdx.x - stride];
        }
        __syncthreads(); /* wait until all threads finish reading 
		                    an element */

        if (threadIdx.x >= stride)
            array[threadIdx.x] += v;

        __syncthreads(); /* wait until all threads finish updating an
		                    element */
     }
     
} // scan_plus




__global__ void caculateWithMatrix(const char  *s1, int n1, const char *s2, int n2,  int *result)
{
     __shared__ int r;
     int value;
     int tid = threadIdx.x;
    if(tid == 0) r = 0;
    __syncthreads();
    if (tid < n1 && tid <n2)
    {
        value = (getScoreFromMatrix (s1[tid] ,s2[tid])); 
        atomicAdd(&r , value);

    }
        

     //scan_plus(flags, BLOCK_DIM);
     //if (tid  == BLOCK_DIM-1; 
     __syncthreads();
     if (tid == 0)
        *result = r;

}
__global__ void caculate(const char  *s1, int n1, const char *s2, int n2,  int *result)
{
    // __shared__ int flags[BLOCK_DIM];
     __shared__ int r;
     int value;
     int tid = threadIdx.x;
    if(tid == 0) r = 0;
    __syncthreads();
    if (tid < n1 && tid <n2)
    {
        value = (s1[tid] == s2[tid]); 
        atomicAdd(&r , value);

    }
        

     //scan_plus(flags, BLOCK_DIM);
     //if (tid  == BLOCK_DIM-1; 
     __syncthreads();
     if (tid == 0)
        *result = r;

}

// returns 0 if successful, otherwise returns 1
// int computeOnGPU(const char *s2) {
//     char *dev_s2;
//     int *dev_result;
//      // null byte at the end is also counted
//     int n2 = strlen(s2)+1;
//     // allocate the memory on the GPU
//     cudaError_t err2 = cudaMalloc((void**)&dev_s2, n2);
//     if (err2 != cudaSuccess)
//     {
//         fprintf(stderr, "CUDA  2 error\n");
//         exit(1);
//     }
//     cudaError_t err3 =  cudaMalloc((void**)&dev_result, sizeof(int));
//     if (err3 != cudaSuccess)
//     {
//         fprintf(stderr, "CUDA 3 error\n");
//         exit(1);
//     }
//     cudaMemcpy(dev_s2, s2, n2, cudaMemcpyHostToDevice);
//     int threadsPerBlock = BLOCK_DIM;
//     int numOfBlocks = 1;
//     //if strlen <1024
//     caculate<<<numOfBlocks, threadsPerBlock>>>(first_str,lenght_first_str,dev_s2, n2, dev_result);
//     err1 = cudaGetLastError();
//     if (err1 != cudaSuccess)
//     {
//         fprintf(stderr , "kerner lanch error\n");
//         exit(1);
//     }
//     // copy the result back from the GPU to the CPU
//     int result;
//     err1 = cudaMemcpy(&result, dev_result, sizeof(int), cudaMemcpyDeviceToHost);		
//     if (err1 != cudaSuccess)
//     {
//         fprintf(stderr , "kerner lanch error");
//         exit(1);
//     }
//     // free memory on the GPU side
//     cudaFree(dev_s1);
//     cudaFree(dev_s2);
//     cudaFree(dev_result);
//     return result;
    
// }
// int computeOnGPUWithMatrix(const char  *s1, const char *s2 ,const int matrix[MATRIX_SIZE][MATRIX_SIZE])
// {
//     cudaMemcpyToSymbol(matrix_caculate, matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
//     char *dev_s1, *dev_s2;
//     int *dev_result;
//     int n1 = strlen(s1)+1; // null byte at the end is also counted
//     int n2 = strlen(s2)+1;

//     // allocate the memory on the GPU
//     cudaError_t err1 = cudaMalloc((void**)&dev_s1, n1);
//     cudaError_t err2 = cudaMalloc((void**)&dev_s2, n2);
//     cudaError_t err3 =  cudaMalloc((void**)&dev_result, sizeof(int));
//     if(err1 != cudaSuccess || err2 != cudaSuccess || err3 != cudaSuccess) {
//         fprintf(stderr, "CUDA error\n");
//         exit(1);
//     }
//     cudaMemcpy(dev_s1, s1, n1, cudaMemcpyHostToDevice);
//     cudaMemcpy(dev_s2, s2, n2, cudaMemcpyHostToDevice);
    
//     int threadsPerBlock = BLOCK_DIM;
//     int numOfBlocks = 1;
 
//     caculateWithMatrix<<<numOfBlocks, threadsPerBlock>>>(dev_s1, n1, dev_s2, n2, dev_result);
 
//     // copy the result back from the GPU to the CPU
//     int result=0;
//     cudaMemcpy(&result, dev_result, sizeof(int), cudaMemcpyDeviceToHost);		
	    
//     // free memory on the GPU side
//     cudaFree(dev_s1);
//     cudaFree(dev_s2);
//     cudaFree(dev_result);

//     return result;
// }
void getFirstStr(char *s1, int n1)
{
    hipMemcpyToSymbol(HIP_SYMBOL(first_str), s1, n1 * sizeof(char));
    hipMemcpyToSymbol(HIP_SYMBOL(length_first_str), &n1, sizeof(int));
}

__global__ void change_offset(char *str, int offset)
{
    int tid = threadIdx.x;
    if (tid < length_first_str)
    {
        str[tid] = first_str[tid + offset];
    }
    if (tid == length_first_str)
    {
        str[tid] = '\0';
    }
}

char *offsetFirstStr(int offset , int lenght)
{
    char *result, *returnStr;
    hipMalloc((void **)&result, lenght);
    int threadsPerBlock = BLOCK_DIM;
    int numOfBlocks = 1;
    change_offset<<<numOfBlocks, threadsPerBlock>>>(result, offset);
    returnStr = (char *)malloc(lenght * sizeof(char));
    hipMemcpy(returnStr, result, lenght * sizeof(char), hipMemcpyDeviceToHost);
    hipFree(result);
    return returnStr;
}

