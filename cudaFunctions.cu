#include <stdio.h>
#include <cstring>
#include <ctype.h>
#include <string.h>
#include <hip/hip_runtime.h>

const int MAX_STRING_SIZE = 3000; // Define MAX_STRING_SIZE as needed
const int MATRIX_SIZE = 26;       // Define MATRIX_SIZE as needed

struct score_alignment
{
    int score;
    int K;
    int off_set;
    char str[MAX_STRING_SIZE];
};
__device__ int device_strlen(const char *str)
{
    int length = 0;
    while (str[length] != '\0')
    {
        length++;
    }
    return length;
}

__device__ void device_strncpy(char *dest, const char *src, int n)
{
    for (int i = 0; i < n; i++)
    {
        dest[i] = src[i];
    }
    dest[n] = '\0';
}

__device__ char gpu_toupper(char c)
{
    if (c >= 'a' && c <= 'A')
        return c - ('a' - 'A');
    return c;
}

__global__ void caculate_result(char *str_to_check, char *first_str, int size_second_str, int *result, int off_set, int *matrix, int k)
{
    __shared__ int r;
    int value;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid == 0)
        r = 0;
    __syncthreads();
    if (tid < size_second_str)
    {
        int i = tid;
        if (tid >= k)
        {

            int x = gpu_toupper(first_str[i + off_set]) - 'A';
            int y = (gpu_toupper(str_to_check[i]) + 1) - 'A';
            value = matrix[x * MATRIX_SIZE + y];
        }
        else
        {
            int x = gpu_toupper(first_str[i + off_set]) - 'A';
            int y = (gpu_toupper(str_to_check[i])) - 'A';
            value = matrix[x * MATRIX_SIZE + y];
        }
        atomicAdd(&r, value);
    }
    __syncthreads();
    if (tid == 0)
        *result = r;
}

int caculate_cuda(const char *str_to_check, const char *first_str, int matrix[MATRIX_SIZE][MATRIX_SIZE])
{

    // Calculate the lengths of the strings
    int size_str_to_check = strlen(str_to_check);
    int size_first_str = strlen(first_str);

    // Check if the lengths exceed the maximum allowed size
    if (size_str_to_check >= MAX_STRING_SIZE || size_first_str >= MAX_STRING_SIZE)
    {
        printf("Error: String length exceeds MAX_STRING_SIZE.\n");
        return -1; // or any appropriate error code
    }

    // Allocate memory on the GPU
    struct score_alignment localMax;
    char *d_str_to_check, *d_first_str;
    int *d_matrix, *dev_result;
    hipMalloc((void **)&d_str_to_check, MAX_STRING_SIZE);
    hipMalloc((void **)&d_first_str, MAX_STRING_SIZE);
    hipMalloc((void **)&d_matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
    hipMalloc((void **)&dev_result, sizeof(int));
    // Copy data from host to device
    hipMemcpy(d_str_to_check, str_to_check, size_str_to_check + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_first_str, first_str, size_first_str + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_matrix, matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(int), hipMemcpyHostToDevice);
    int threadsPerBlock = 256;
    int result = 0;
    int blocksPerGrid = (size_str_to_check > MAX_STRING_SIZE) ? size_str_to_check / threadsPerBlock : 1;

    int max_score = 0;
    int sqn_taries = (size_str_to_check < size_first_str) ? (size_first_str - size_str_to_check) : (size_str_to_check - size_first_str);

    for (int off_set = 0; off_set <= sqn_taries; off_set++)
    {
        for (int k = 0; k < size_str_to_check; k++)
        {
            caculate_result<<<blocksPerGrid, threadsPerBlock>>>(
                d_str_to_check, d_first_str, size_str_to_check,
                dev_result, off_set, d_matrix, k);
            hipMemcpy(&result, dev_result, sizeof(int), hipMemcpyDeviceToHost);
            if (result > max_score)
            {
                max_score = result;
                localMax.score = result;
                localMax.K = k;
                localMax.off_set = off_set;
            }
        }
    }
    // Define block and grid dimensions

    // Launch the CUDA kernel

    // Copy the result back from the device to the host

    // Free allocated memory on the device
    hipFree(d_str_to_check);
    hipFree(d_first_str);
    hipFree(d_matrix);
    hipFree(dev_result);
    // Print the result
    printf("\nmy_rank [0] For the string %s,\n", str_to_check);
    printf("We found that the max score alignment %d is from K - %d and off set - %d\n", localMax.score, localMax.K, localMax.off_set);

    return 0;
}
