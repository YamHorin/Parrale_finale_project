#include <stdio.h>
#include <cstring>
#include <ctype.h>
#include <string.h>
#include <hip/hip_runtime.h> 

const int MAX_STRING_SIZE = 3000; // Define MAX_STRING_SIZE as needed
const int MATRIX_SIZE = 26;       // Define MATRIX_SIZE as needed

struct score_alignment {
    int score;
    int K;
    int off_set;
    char str[MAX_STRING_SIZE];
};
__device__ int device_strlen(const char* str) {
    int length = 0;
    while (str[length] != '\0') {
        length++;
    }
    return length;
}

__device__ void device_strncpy(char* dest, const char* src, int n) {
    for (int i = 0; i < n; i++) {
        dest[i] = src[i];
    }
    dest[n] = '\0';
}

__device__ char gpu_toupper(char c)
{
    if (c >= 'a' && c <= 'A')
        return c - ('a' - 'A');
    return c;
}

__device__ int caculate_result_without_matrix(const char *s2, int off_set, const char *first_str) {
    int length = device_strlen(s2);
    int result = 0;
    
    for (int i = 0; i < length; i++) {
        if (first_str[i + off_set] == s2[i]) {
            result++;
        }
    }
    
    return result;
}

__device__ int calculate_result_with_matrix(const char *s2, int *matrix, int off_set, const char *first_str) {
    int length = device_strlen(s2);
    int result = 0;
    
    for (int i = 0; i < length; i++) {
        int x = first_str[i + off_set] - 'A';
        int y = s2[i] - 'A';
        
        if (x < 0 || x >= MATRIX_SIZE || y < 0 || y >= MATRIX_SIZE) {
            // Handle out-of-bounds characters.
            return -1; // or any appropriate error code
        }
        result += matrix[x * MATRIX_SIZE + y];
    }
    
    return result;
}

__device__ void Mutanat_Squence2(char *str, int k, int size_str) {
    for (int i = k; i <= size_str; i++) {
        if (gpu_toupper(str[i]) >= 'Z') {
            str[i] = 'A';
        }
        if (i == size_str) {
            str[i] = '\0';
        } else {
            str[i] = gpu_toupper(str[i] + 1);
        }
    }
}
__global__ void caculate_result(char *str_to_check, char *first_str, int size_second_str,  int *result , int off_set
                                , int* matrix ,int k)
{
     __shared__ int r;
     int value;
     int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid == 0) r = 0;
    __syncthreads();
    if (tid <size_second_str)
    {
        int i  = tid;
        if (tid>=k)
        {
            
            int x = gpu_toupper(first_str[i + off_set]) - 'A';
            int y = (gpu_toupper(str_to_check[i])+1) - 'A';
            value = matrix[x * MATRIX_SIZE + y];
        }
        else
        {
            int x = gpu_toupper(first_str[i + off_set]) - 'A';
            int y = (gpu_toupper(str_to_check[i])) - 'A';
            value = matrix[x * MATRIX_SIZE + y];
        }
        atomicAdd(&r , value);

    }
     __syncthreads();
     if (tid == 0)
        *result = r;

}
// __global__ void cuda_caculate_max_score(char *str_to_check, char *first_str, int how_to_caculate,
//                                         int *matrix, score_alignment *localMax) {
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
//     if (tid < sqn_taries * size_str_to_check) {
//         int off_set = tid / size_str_to_check;
//         int k = tid % size_str_to_check;

//         // Create a copy of str_to_check
//         char mutated_str[MAX_STRING_SIZE];
//         device_strncpy(mutated_str, str_to_check, MAX_STRING_SIZE - 1);
//         mutated_str[MAX_STRING_SIZE - 1] = '\0';

//         // Mutate the sequence
//         Mutanat_Squence2(mutated_str, k, size_str_to_check);

//         int score = 0;

//         if (how_to_caculate == 0) {
//             score = caculate_result_without_matrix(mutated_str, off_set, first_str);
//         } else {
//             score = calculate_result_with_matrix(mutated_str, matrix, off_set, first_str);
//         }

//         // Update localMax if a higher score is found
//         atomicMax(&localMax->score, score);
//         if (score == localMax->score) {
//             localMax->K = k;
//             localMax->off_set = off_set;
//         }
//     }
// }

int caculate_cuda(const char *str_to_check, const char *first_str, int matrix[MATRIX_SIZE][MATRIX_SIZE]) {


    // Calculate the lengths of the strings
    int size_str_to_check = strlen(str_to_check);
    int size_first_str = strlen(first_str);

    // Check if the lengths exceed the maximum allowed size
    if (size_str_to_check >= MAX_STRING_SIZE || size_first_str >= MAX_STRING_SIZE) {
        printf("Error: String length exceeds MAX_STRING_SIZE.\n");
        return -1; // or any appropriate error code
    }

    // Allocate memory on the GPU
    struct score_alignment localMax;
    char *d_str_to_check, *d_first_str;
    int *d_matrix , *dev_result;
    hipMalloc((void**)&d_str_to_check, MAX_STRING_SIZE);
    hipMalloc((void**)&d_first_str, MAX_STRING_SIZE);
    hipMalloc((void**)&d_matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
    hipMalloc((void**)&dev_result, sizeof(int));
    // Copy data from host to device
    hipMemcpy(d_str_to_check, str_to_check, size_str_to_check + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_first_str, first_str, size_first_str + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_matrix, matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(int), hipMemcpyHostToDevice);
    int threadsPerBlock = 256;
    int result=0;
    int blocksPerGrid = (size_str_to_check > MAX_STRING_SIZE)? size_str_to_check/threadsPerBlock : 1;

    int max_score = 0;
    int sqn_taries = (size_str_to_check < size_first_str) ? (size_first_str - size_str_to_check) : (size_str_to_check - size_first_str);

    for (int off_set = 0; off_set <= sqn_taries; off_set++)
    {
        for (int k = 0; k < size_str_to_check; k++)
        {
               caculate_result <<<blocksPerGrid, threadsPerBlock>>>(
                d_str_to_check,d_first_str,  size_str_to_check,  
                dev_result,  off_set,  d_matrix , k);
            hipMemcpy(&result, dev_result, sizeof(int), hipMemcpyDeviceToHost);
            if (result >max_score)
            {
                max_score = result;
                localMax.score = result;
                localMax.K = k;
                localMax.off_set =  off_set;
            }


        }
    }
    // Define block and grid dimensions
    

    // Launch the CUDA kernel

    // Copy the result back from the device to the host

    // Free allocated memory on the device
    hipFree(d_str_to_check);
    hipFree(d_first_str);
    hipFree(d_matrix);
    hipFree(dev_result);
    // Print the result
    printf("\nFor the string %s,\n", str_to_check);
    printf("We found that the max score alignment %d is from K - %d and off set - %d\n", localMax.score, localMax.K, localMax.off_set);

    return localMax.score;
}

/*
for the string �Ss[� 
, we found that the max score alignment 22069 is from K  - 5 and off set - 0 

*/