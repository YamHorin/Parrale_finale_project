#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <ctype.h>
#include <hip/hip_runtime.h> 
const int MAX_STRING_SIZE = 3000; // Define MAX_STRING_SIZE as needed
const int MATRIX_SIZE = 26;     // Define MATRIX_SIZE as needed



__device__ int caculate_result_without_matrix(const char *s2, int off_set, const char *first_str) {
    int length = strlen(s2);
    int result = 0;
    
    for (int i = 0; i < length; i++) {
        if (first_str[i + off_set] == s2[i]) {
            result++;
        }
    }
    
    return result;
}

__device__ int calculate_result_with_matrix(const char *s2, int *matrix, int off_set, const char *first_str) {
    int length = strlen(s2);
    int result = 0;
    
    for (int i = 0; i < length; i++) {
        int x = first_str[i + off_set] - 'A';
        int y = s2[i] - 'A';
        
        if (x < 0 || x >= MATRIX_SIZE || y < 0 || y >= MATRIX_SIZE) {
            // Handle out-of-bounds characters.
            return -1; // or any appropriate error code
        }
        result += matrix[x * MATRIX_SIZE + y];
    }
    
    return result;
}

__device__ void Mutanat_Squence(char *str, int k, int size_str) {
    for (int i = k; i <= size_str; i++) {
        if (toupper(str[i]) >= 'Z') {
            str[i] = 'A';
        }
        if (i == size_str) {
            str[i] = '\0';
        } else {
            str[i] = toupper(str[i] + 1);
        }
    }
}



__global__ void cuda_caculate_max_score(char *str_to_check, char *first_str, int how_to_caculate,
                                        int *matrix, score_alignment2 *localMax) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int size_str_to_check = strlen(str_to_check);
    int length_first_str = strlen(first_str);
    int sqn_taries = (size_str_to_check < length_first_str) ? (length_first_str - size_str_to_check) : (size_str_to_check - length_first_str);
    
    if (tid < sqn_taries * size_str_to_check) {
        int off_set = tid / size_str_to_check;
        int k = tid % size_str_to_check;

        // Create a copy of str_to_check
        char mutated_str[MAX_STRING_SIZE];
        strncpy(mutated_str, str_to_check, MAX_STRING_SIZE - 1);
        mutated_str[MAX_STRING_SIZE - 1] = '\0';

        // Mutate the sequence
        Mutanat_Squence(mutated_str, k, size_str_to_check);

        int score = 0;

        if (how_to_caculate == 0) {
            score = caculate_result_without_matrix(mutated_str, off_set, first_str);
        } else {
            score = calculate_result_with_matrix(mutated_str, matrix, off_set, first_str);
        }

        // Update localMax if a higher score is found
        atomicMax(&localMax->score, score);
        if (score == localMax->score) {
            localMax->K = k;
            localMax->off_set = off_set;
        }
    }
}

/*
cudaFunctions2.cu(57): error: identifier "score_alignment2" is undefined

cudaFunctions2.cu(105): error: identifier "score_alignment" is undefined

cudaFunctions2.cu(113): error: incomplete type is not allowed

cudaFunctions2.cu(117): error: incomplete type is not allowed



*/



int caculate_cuda(char *str_to_check, char *first_str ,int matrix[MATRIX_SIZE][MATRIX_SIZE]) {
    // Initialize data and matrices here
    int how_to_caculate = 0; // Set to 0 for NO_MATRIX_SCORE or 1 for MATRIX_SCORE

    // Allocate memory on the GPU
    char *d_str_to_check, *d_first_str;
    int *d_matrix;
    struct cscore_alignment2 *d_localMax;
    hipMalloc((void**)&d_str_to_check, MAX_STRING_SIZE);
    hipMalloc((void**)&d_first_str, MAX_STRING_SIZE);
    hipMalloc((void**)&d_matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
    hipMalloc((void**)&d_localMax, sizeof(score_alignment));

    // Copy data from host to device
    hipMemcpy(d_str_to_check, str_to_check, MAX_STRING_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_first_str, first_str, MAX_STRING_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(d_matrix, matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(int), hipMemcpyHostToDevice);
    
    // Initialize localMax on the host and copy it to the device
    struct score_alignment2 localMax;
    localMax.score = 0;
    localMax.K = 0;
    localMax.off_set = 0;
    hipMemcpy(d_localMax, &localMax, sizeof(score_alignment2), hipMemcpyHostToDevice);

    // Define block and grid dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (MAX_STRING_SIZE * MAX_STRING_SIZE + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the CUDA kernel
    cuda_caculate_max_score<<<blocksPerGrid, threadsPerBlock>>>(d_str_to_check, d_first_str, how_to_caculate, d_matrix, d_localMax);

    // Copy the result back from the device to the host
    hipMemcpy(&localMax, d_localMax, sizeof(score_alignment), hipMemcpyDeviceToHost);

    // Free allocated memory on the device
    hipFree(d_str_to_check);
    hipFree(d_first_str);
    hipFree(d_matrix);
    hipFree(d_localMax);

    // Print the result
    printf("\nFor the string %s,\n", str_to_check);
    printf("We found that the max score alignment %d is from K - %d and off set - %d\n", localMax.score, localMax.K, localMax.off_set);

    return localMax.score;
}