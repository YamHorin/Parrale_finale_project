#include <stdio.h>
#include <cstring>
#include <ctype.h>
#include <string.h>
#include <hip/hip_runtime.h> 
const int MAX_STRING_SIZE = 3000; // Define MAX_STRING_SIZE as needed
const int MATRIX_SIZE = 26;       // Define MATRIX_SIZE as needed

struct score_alignment {
    int score;
    int K;
    int off_set;
    char str[MAX_STRING_SIZE];
};
__device__ int device_strlen(const char* str) {
    int length = 0;
    while (str[length] != '\0') {
        length++;
    }
    return length;
}

__device__ void device_strncpy(char* dest, const char* src, int n) {
    for (int i = 0; i < n; i++) {
        dest[i] = src[i];
    }
    dest[n] = '\0';
}

__device__ char gpu_toupper(char c)
{
    if (c >= 'a' && c <= 'A')
        return c - ('a' - 'A');
    return c;
}

__device__ int caculate_result_without_matrix(const char *s2, int off_set, const char *first_str) {
    int length = device_strlen(s2);
    int result = 0;
    
    for (int i = 0; i < length; i++) {
        if (first_str[i + off_set] == s2[i]) {
            result++;
        }
    }
    
    return result;
}

__device__ int calculate_result_with_matrix(const char *s2, int *matrix, int off_set, const char *first_str) {
    int length = device_strlen(s2);
    int result = 0;
    
    for (int i = 0; i < length; i++) {
        int x = first_str[i + off_set] - 'A';
        int y = s2[i] - 'A';
        
        if (x < 0 || x >= MATRIX_SIZE || y < 0 || y >= MATRIX_SIZE) {
            // Handle out-of-bounds characters.
            return -1; // or any appropriate error code
        }
        result += matrix[x * MATRIX_SIZE + y];
    }
    
    return result;
}

__device__ void Mutanat_Squence2(char *str, int k, int size_str) {
    for (int i = k; i <= size_str; i++) {
        if (gpu_toupper(str[i]) >= 'Z') {
            str[i] = 'A';
        }
        if (i == size_str) {
            str[i] = '\0';
        } else {
            str[i] = gpu_toupper(str[i] + 1);
        }
    }
}

__global__ void cuda_caculate_max_score(char *str_to_check, char *first_str, int how_to_caculate,
                                        int *matrix, score_alignment *localMax) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int size_str_to_check = device_strlen(str_to_check);
    int length_first_str = device_strlen(first_str);
    int sqn_taries = (size_str_to_check < length_first_str) ? (length_first_str - size_str_to_check) : (size_str_to_check - length_first_str);
    
    if (tid < sqn_taries * size_str_to_check) {
        int off_set = tid / size_str_to_check;
        int k = tid % size_str_to_check;

        // Create a copy of str_to_check
        char mutated_str[MAX_STRING_SIZE];
        device_strncpy(mutated_str, str_to_check, MAX_STRING_SIZE - 1);
        mutated_str[MAX_STRING_SIZE - 1] = '\0';

        // Mutate the sequence
        Mutanat_Squence2(mutated_str, k, size_str_to_check);

        int score = 0;

        if (how_to_caculate == 0) {
            score = caculate_result_without_matrix(mutated_str, off_set, first_str);
        } else {
            score = calculate_result_with_matrix(mutated_str, matrix, off_set, first_str);
        }

        // Update localMax if a higher score is found
        atomicMax(&localMax->score, score);
        if (score == localMax->score) {
            localMax->K = k;
            localMax->off_set = off_set;
        }
    }
}

int caculate_cuda(const char *str_to_check, const char *first_str, int matrix[MATRIX_SIZE][MATRIX_SIZE]) {
    // Initialize data and matrices here
    int how_to_caculate = 0; // Set to 0 for NO_MATRIX_SCORE or 1 for MATRIX_SCORE

    // Calculate the lengths of the strings
    int size_str_to_check = strlen(str_to_check);
    int size_first_str = strlen(first_str);

    // Check if the lengths exceed the maximum allowed size
    if (size_str_to_check >= MAX_STRING_SIZE || size_first_str >= MAX_STRING_SIZE) {
        printf("Error: String length exceeds MAX_STRING_SIZE.\n");
        return -1; // or any appropriate error code
    }

    // Allocate memory on the GPU
    char *d_str_to_check, *d_first_str;
    int *d_matrix;
    score_alignment *d_localMax;
    hipMalloc((void**)&d_str_to_check, MAX_STRING_SIZE);
    hipMalloc((void**)&d_first_str, MAX_STRING_SIZE);
    hipMalloc((void**)&d_matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
    hipMalloc((void**)&d_localMax, sizeof(score_alignment));

    // Copy data from host to device
    hipMemcpy(d_str_to_check, str_to_check, size_str_to_check + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_first_str, first_str, size_first_str + 1, hipMemcpyHostToDevice);
    hipMemcpy(d_matrix, matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(int), hipMemcpyHostToDevice);
    
    // Initialize localMax on the host and copy it to the device
    score_alignment localMax;
    localMax.score = 0;
    localMax.K = 0;
    localMax.off_set = 0;
    hipMemcpy(d_localMax, &localMax, sizeof(score_alignment), hipMemcpyHostToDevice);




    /*
    to do 
    two for loops
    one for k 
    the other on for off set

    global caculate that will get str 1 , str2 , k, off_set 
    caculation is simular to before 
    max chaeck in this func , not the global 

    */
    // Define block and grid dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (size_str_to_check * (size_first_str - size_str_to_check) + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the CUDA kernel
    cuda_caculate_max_score<<<blocksPerGrid, threadsPerBlock>>>(d_str_to_check, d_first_str, how_to_caculate, d_matrix, d_localMax);

    // Copy the result back from the device to the host
    hipMemcpy(&localMax, d_localMax, sizeof(score_alignment), hipMemcpyDeviceToHost);

    // Free allocated memory on the device
    hipFree(d_str_to_check);
    hipFree(d_first_str);
    hipFree(d_matrix);
    hipFree(d_localMax);

    // Print the result
    printf("\nFor the string %s,\n", str_to_check);
    printf("We found that the max score alignment %d is from K - %d and off set - %d\n", localMax.score, localMax.K, localMax.off_set);

    return localMax.score;
}

/*
for the string �Ss[� 
, we found that the max score alignment 22069 is from K  - 5 and off set - 0 

*/